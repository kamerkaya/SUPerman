#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include "flags.h"
#include "gpu_wrappers.h"

static int glob_nov;
static int glob_sizeof_c;
static int glob_sizeof_s;

//This is a CPU helper kernel for hybrid setting
template <class T>
double cpu_perman64(T* mat_t,
		    double x[],
		    int nov,
		    long long start,
		    long long end,
		    int threads) {
  double p = 0; //product of the elements in vector 'x'
  long long one = 1;
  long long chunk_size = (end - start) / threads + 1;
  omp_set_num_threads(threads);

  #pragma omp parallel
  { 
    double my_x[nov];
    for (int i = 0; i < nov; i++) {
      my_x[i] = x[i];
    }
    int tid = omp_get_thread_num();
    long long my_start = start + tid * chunk_size;
    long long my_end = min(start + ((tid+1) * chunk_size), end);
    
    double *xptr; 
    int s;  //+1 or -1 
    double prod; //product of the elements in vector 'x'
    double my_p = 0;
    long long i = my_start;
    long long gray = (i-1) ^ ((i-1) >> 1);

    for (int k = 0; k < (nov-1); k++) {
      if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
        xptr = (double*)my_x;
        for (int j = 0; j < nov; j++) {
          *xptr += mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
          xptr++;
        }
      }
    }
    int k;

    int prodSign = 1;
    if(i & 1LL) {
      prodSign = -1;
    }
    while (i < my_end) {
      //compute the gray code
      k = __builtin_ctzll(i);
      gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
      //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
      s = ((one << k) & gray) ? 1 : -1;
      
      prod = 1.0;
      xptr = (double*)my_x;
      for (int j = 0; j < nov; j++) {
        *xptr += s * mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
        prod *= *xptr++;  //product of the elements in vector 'x'
      }

      my_p += prodSign * prod; 
      prodSign *= -1;
      i++;
    }

    #pragma omp atomic
      p += my_p;
  }

  return p;
}

int xshared_sharedmem(int b){
  return glob_nov*b*glob_sizeof_c;
}

//Same with above but lets keep it just to prevent confusion
int xshared_coalescing_sharedmem(int b){ 
  return glob_nov*b*glob_sizeof_c;
}

int xshared_coalescing_mshared_sharedmem(int b){
  return (glob_nov*b*glob_sizeof_c + glob_nov*glob_nov*glob_sizeof_s);
}

template <class C, class S>
__global__ void kernel_xglobal(S* mat_t,
			       C* x,
			       C* p,
			       int nov) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  
  long long number_of_threads = blockDim.x * gridDim.x;

  long long one = 1;
  long long start = 1;
  long long end = (1LL << (nov-1));
  
  long long chunk_size = (end-start) / number_of_threads + 1; //Is this the problem

  long long my_start = start + tid * chunk_size;
  long long my_end = min(start + ((tid+1) * chunk_size), end);
     
  C s;  //+1 or -1 
  C prod; //product of the elements in vector 'x'
  C my_p = 0;
  long long i = my_start;
  long long gray = (i-1) ^ ((i-1) >> 1);

  for (int k = 0; k < (nov-1); k++) {
    if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
      for (int j = 0; j < nov; j++) {
        x[tid*nov + j] += mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      }
    }
  }
    
  long long gray_diff;
  int k;

  int prodSign = 1;
  if(i & 1LL) {
    prodSign = -1;
  }

  while (i < my_end) {
    gray_diff = (i ^ (i >> 1)) ^ gray;
    k = __ffsll(gray_diff) - 1;
    gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
    //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
    s = ((one << k) & gray) ? 1 : -1;
      
    prod = 1.0;
    for (int j = 0; j < nov; j++) {
      x[tid*nov + j] += s * mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      prod *= x[tid*nov + j];  //product of the elements in vector 'x'
    }

    my_p += prodSign * prod; 
    prodSign *= -1;
    i++;
  }

  p[tid] = my_p;
}

template <class C, class S>
__global__ void kernel_xlocal(S* mat_t, C* x, C* p, int nov) {

  C my_x[40]; //Again, it is problematic for matrices > 40 but anyways, we will not calculate them with this kernel. Another problem is, this may cause register spilling with different GPUs.
  
  for (int k = 0; k < nov; k++) {
    my_x[k] = x[k];
  }
  
  long long number_of_threads = blockDim.x * gridDim.x;

  long long one = 1;
  long long start = 1;
  long long end = (1LL << (nov-1));
  
  long long chunk_size = (end - start) / number_of_threads + 1;

  
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  long long my_start = start + tid * chunk_size;
  long long my_end = min(start + ((tid+1) * chunk_size), end);
    
  C *xptr; 
  C s;  //+1 or -1 
  C prod; //product of the elements in vector 'x'
  C my_p = 0;
  long long i = my_start;
  long long gray = (i-1) ^ ((i-1) >> 1);

  for (int k = 0; k < (nov-1); k++) {
    if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
      xptr = (C*)my_x;
      for (int j = 0; j < nov; j++) {
        *xptr += mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
        xptr++;
      }
    }
  }
    
  long long gray_diff;
  int k;

  int prodSign = 1;
  if(i & 1LL) {
    prodSign = -1;
  }

  while (i < my_end) {
    gray_diff = (i ^ (i >> 1)) ^ gray;
    k = __ffsll(gray_diff) - 1;
    gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
    //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
    s = ((one << k) & gray) ? 1 : -1;
      
    prod = 1.0;
    xptr = (C*)my_x;
    for (int j = 0; j < nov; j++) {
      *xptr += s * mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      prod *= *xptr++;  //product of the elements in vector 'x'
    }

    my_p += prodSign * prod; 
    prodSign *= -1;
    i++;
  }

  p[tid] = my_p;
}

template <class C, class S>
__global__ void kernel_xshared(S* mat_t, C* x, C* p, int nov) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;

  extern __shared__ double shared_mem[]; 
  C *my_x = (C*)shared_mem; // size = nov * BLOCK_SIZE

  for (int k = 0; k < nov; k++) {
    my_x[thread_id*nov + k] = x[k];
  }
  
  long long number_of_threads = blockDim.x * gridDim.x;

  long long one = 1;
  long long start = 1;
  long long end = (1LL << (nov-1));
  
  long long chunk_size = (end - start) / number_of_threads + 1;

  long long my_start = start + tid * chunk_size;
  long long my_end = min(start + ((tid+1) * chunk_size), end);
     
  C s;  //+1 or -1 
  C prod; //product of the elements in vector 'x'
  C my_p = 0;
  long long i = my_start;
  long long gray = (i-1) ^ ((i-1) >> 1);

  for (int k = 0; k < (nov-1); k++) {
    if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
      for (int j = 0; j < nov; j++) {
        my_x[thread_id*nov + j] += mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      }
    }
  }
    
  long long gray_diff;
  int k;

  int prodSign = 1;
  if(i & 1LL) {
    prodSign = -1;
  }

  while (i < my_end) {
    gray_diff = (i ^ (i >> 1)) ^ gray;
    k = __ffsll(gray_diff) - 1;
    gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
    //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
    s = ((one << k) & gray) ? 1 : -1;
      
    prod = 1.0;
    for (int j = 0; j < nov; j++) {
      my_x[thread_id*nov + j] += s * mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      prod *= my_x[thread_id*nov + j];  //product of the elements in vector 'x'
    }

    my_p += prodSign * prod; 
    prodSign *= -1;
    i++;
  }

  p[tid] = my_p;
}

template <class C, class S>
__global__ void kernel_xshared_coalescing(S* mat_t, C* x, C* p, int nov) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;
  int block_dim = blockDim.x;

  extern __shared__ double shared_mem[]; 
  C *my_x = (C*)shared_mem; // size = nov * BLOCK_SIZE

  for (int k = 0; k < nov; k++) {
    my_x[block_dim*k + thread_id] = x[k];
  }

  long long number_of_threads = blockDim.x * gridDim.x;

  long long one = 1;
  long long start = 1;
  long long end = (1LL << (nov-1));
  
  long long chunk_size = (end - start) / number_of_threads + 1;

  long long my_start = start + tid * chunk_size;
  long long my_end = min(start + ((tid+1) * chunk_size), end);
  
  C s;  //+1 or -1 
  C prod; //product of the elements in vector 'x'
  C my_p = 0;
  long long i = my_start;
  long long gray = (i-1) ^ ((i-1) >> 1);

  for (int k = 0; k < (nov-1); k++) {
    if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
      for (int j = 0; j < nov; j++) {
        my_x[block_dim*j + thread_id] += mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      }
    }
  }
  
  long long gray_diff;
  int k;

  int prodSign = 1;
  if(i & 1LL) {
    prodSign = -1;
  }

  while (i < my_end) {
    gray_diff = (i ^ (i >> 1)) ^ gray;
    k = __ffsll(gray_diff) - 1;
    gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
    //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
    s = ((one << k) & gray) ? 1 : -1;
      
    prod = 1.0;
    for (int j = 0; j < nov; j++) {
      my_x[block_dim*j + thread_id] += s * mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      prod *= my_x[block_dim*j + thread_id];  //product of the elements in vector 'x'
    }

    my_p += prodSign * prod; 
    prodSign *= -1;
    i++;
  }

  p[tid] = my_p;
}

template <class C, class S>
__global__ void kernel_xshared_coalescing_mshared(S* mat_t, C* x, C* p, int nov, long long start, long long end) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int thread_id = threadIdx.x;
  int block_dim = blockDim.x;

  extern __shared__ double shared_mem[]; 
  C *my_x = (C*)shared_mem; // size = nov * BLOCK_SIZE
  S *shared_mat_t = (S*) &my_x[nov * block_dim]; // size = nov * nov

  for (int k = 0; k < nov; k++) {
    my_x[block_dim*k + thread_id] = x[k];
  }

  for (int k = 0; k < ((nov*nov)/block_dim + 1); k++) {
    if ((block_dim * k + thread_id) < (nov * nov))
    shared_mat_t[block_dim * k + thread_id] = mat_t[block_dim * k + thread_id];
  }

  __syncthreads();

  long long number_of_threads = blockDim.x * gridDim.x;

  long long one = 1;
  
  long long chunk_size = (end - start) / number_of_threads + 1;

  long long my_start = start + tid * chunk_size;
  long long my_end = min(start + ((tid+1) * chunk_size), end);
  
  C s;  //+1 or -1 
  C prod; //product of the elements in vector 'x'
  C my_p = 0;
  long long i = my_start;
  long long gray = (i-1) ^ ((i-1) >> 1);

  for (int k = 0; k < (nov-1); k++) {
    if ((gray >> k) & 1LL) { // whether kth column should be added to x vector or not
      for (int j = 0; j < nov; j++) {
        my_x[block_dim*j + thread_id] += shared_mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      }
    }
  }
    
  long long gray_diff;
  int k;

  int prodSign = 1;
  if (i & 1LL) {
    prodSign = -1;
  }
  while (i < my_end) {
    gray_diff = (i ^ (i >> 1)) ^ gray;
    k = __ffsll(gray_diff) - 1;
    gray ^= (one << k); // Gray-code order: 1,3,2,6,7,5,4,12,13,15,...
    //decide if subtract of not - if the kth bit of gray is one then 1, otherwise -1
    s = ((one << k) & gray) ? 1 : -1;
      
    prod = 1.0;
    for (int j = 0; j < nov; j++) {
      my_x[block_dim*j + thread_id] += s * shared_mat_t[(k * nov) + j]; // see Nijenhuis and Wilf - update x vector entries
      prod *= my_x[block_dim*j + thread_id];  //product of the elements in vector 'x'
    }

    my_p += prodSign * prod; 
    prodSign *= -1;
    i++;
  }

  p[tid] = my_p;
}

template <class C, class S>
extern Result gpu_perman64_xglobal(DenseMatrix<S>* densemat, flags flags) {

  //Pack parameters
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters

  //Pack flags//
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//

  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();

  hipOccupancyMaxPotentialBlockSize(&grid_dim,
                                     &block_dim,
                                     &kernel_xglobal<C,S>,
                                     0,
                                     0);
  
  printf("==SC== No Shared memory is used for the kernel..\n");
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);

  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }
  
  
  C x[nov]; 
  C rs; //row sum
  C p = 1; //product of the elements in vector 'x'
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //create the transpose of the matrix
  S* mat_t = new S[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  C *h_x = new C[nov*grid_dim*block_dim];
  for (int i = 0; i < nov*grid_dim*block_dim; i++) {
    h_x[i] = x[i%nov];
  }
  
  S *d_mat_t;
  C *d_x;
  C *d_p;
  C *h_p = new C[grid_dim * block_dim];

  hipMalloc( &d_x, (nov*grid_dim*block_dim) * sizeof(C));
  hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat_t, (nov * nov) * sizeof(S));

  hipMemcpy( d_x, h_x, (nov*grid_dim*block_dim) * sizeof(C), hipMemcpyHostToDevice);
  hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);

  //double stt = omp_get_wtime();
  kernel_xglobal<C,S><<<grid_dim , block_dim>>>(d_mat_t, d_x, d_p, nov);
  hipDeviceSynchronize();
  //double enn = omp_get_wtime();
  //printf("Kernel in %f \n", enn - stt);
  //cout << "kernel" << " in " << (enn - stt) << endl;
  
  hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);

  hipFree(d_mat_t);
  hipFree(d_x);
  hipFree(d_p);

  double return_p = 0;
  
  for (int i = 0; i < grid_dim * block_dim; i++) {
    return_p += (double)h_p[i];
  }

  delete [] mat_t;
  delete [] h_x;
  delete [] h_p;

  double perman = (4*(nov&1)-2) * return_p;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;

  
  //return((4*(nov&1)-2) * p);
}

template <class C, class S>
  extern Result gpu_perman64_xlocal(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//
  
  //Pack flags//
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;
  int grid_dim_multip = flags.grid_multip;
  int device_id = flags.device_id;
  //Pack flags//
  
  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();
  
  hipOccupancyMaxPotentialBlockSize(&grid_dim,
                                     &block_dim,
                                     &kernel_xlocal<C,S>,
                                     0,
                                     0);
  
  printf("==SC== No Shared memory is used for the kernel..\n");
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);
  
  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }
  
  C x[nov]; 
  C rs; //row sum
  C p = 1; //product of the elements in vector 'x'
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }
  
  //create the transpose of the matrix
  S* mat_t = new S[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }
  
  S *d_mat_t;
  C *d_x, *d_p;
  C *h_p = new C[grid_dim * block_dim];
  
  hipMalloc( &d_x, (nov) * sizeof(C));
  hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat_t, (nov * nov) * sizeof(S));

  hipMemcpy( d_x, x, (nov) * sizeof(C), hipMemcpyHostToDevice);
  hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);
  
  //double stt = omp_get_wtime();
  kernel_xlocal<C,S><<<grid_dim , block_dim>>> (d_mat_t, d_x, d_p, nov);
  hipDeviceSynchronize();
  //double enn = omp_get_wtime();
  //printf("Kernel in %f \n", enn - stt);
  //cout << "kernel" << " in " << (enn - stt) << endl;
  
  hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);
  
  hipFree(d_mat_t);
  hipFree(d_x);
  hipFree(d_p);

  double return_p = p;
  
  for (int i = 0; i < grid_dim * block_dim; i++) {
    return_p += (double)h_p[i];
  }
  
  delete[] mat_t;
  delete[] h_p;

  double perman = (4*(nov&1)-2) * return_p;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;
  
  //return((4*(nov&1)-2) * p);
}

template <class C, class S>
  extern Result gpu_perman64_xshared(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters

  //Pack flags//
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//

  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();

  C x[nov]; 
  C rs; //row sum
  C p = 1; //product of the elements in vector 'x'
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //For variable smem
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  //For variable smem

  hipOccupancyMaxPotentialBlockSizeVariableSMem(&grid_dim,
                                                 &block_dim,
                                                 &kernel_xshared<C,S>,
                                                 xshared_sharedmem,
                                                 0);

  size_t size = nov*block_dim*sizeof(C);
  
  printf("==SC== Shared memory per block is set to : %zu \n", size);
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);

  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }


  //create the transpose of the matrix
  S* mat_t = new S[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }
  
  S *d_mat_t;
  C *d_x, *d_p;
  C *h_p = new C[grid_dim * block_dim];

  hipMalloc( &d_x, (nov) * sizeof(C));
  hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat_t, (nov * nov) * sizeof(S));

  hipMemcpy( d_x, x, (nov) * sizeof(C), hipMemcpyHostToDevice);
  hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);
  
  //double stt = omp_get_wtime();
  kernel_xshared<C,S><<<grid_dim , block_dim , size>>> (d_mat_t, d_x, d_p, nov);
  hipDeviceSynchronize();
  //double enn = omp_get_wtime();
  //printf("Kernel in %f \n", enn - stt);
  //cout << "kernel" << " in " << (enn - stt) << endl;
  
  hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);

  hipFree(d_mat_t);
  hipFree(d_x);
  hipFree(d_p);

  double return_p = p;
  
  for (int i = 0; i < grid_dim * block_dim; i++) {
    return_p += (double)h_p[i];
  }

  delete [] mat_t;
  delete[] h_p;

  double perman = (4*(nov&1)-2) * return_p;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;
  
  //return((4*(nov&1)-2) * p);
}

template <class C, class S>
extern Result gpu_perman64_xshared_coalescing(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//

  //Pack flags//
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags//

  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();
  
  C x[nov]; 
  C rs; //row sum
  C p = 1; //product of the elements in vector 'x'
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //For variable smem
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  //For variable smem

  hipOccupancyMaxPotentialBlockSizeVariableSMem(&grid_dim,
                                                 &block_dim,
                                                 &kernel_xshared_coalescing<C,S>,
                                                 xshared_coalescing_sharedmem,
                                                 0);

  size_t size = nov*block_dim*sizeof(C);
  
  printf("==SC== Shared memory per block is set to : %zu \n", size);
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);
  
  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }
  
  //create the transpose of the matrix
  S* mat_t = new S[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  
  S *d_mat_t;
  C *d_x, *d_p;
  C *h_p = new C[grid_dim * block_dim];

  hipMalloc( &d_x, (nov) * sizeof(C));
  hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat_t, (nov * nov) * sizeof(S));

  hipMemcpy( d_x, x, (nov) * sizeof(C), hipMemcpyHostToDevice);
  hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);

  //double stt = omp_get_wtime();
  kernel_xshared_coalescing<C,S><<<grid_dim , block_dim , size>>> (d_mat_t, d_x, d_p, nov);
  hipDeviceSynchronize();
  //double enn = omp_get_wtime();
  //printf("Kernel in %f \n", enn - stt);
  //cout << "kernel" << " in " << (enn - stt) << endl;
  
  hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);

  hipFree(d_mat_t);
  hipFree(d_x);
  hipFree(d_p);

  double return_p = p;
  
  for (int i = 0; i < grid_dim * block_dim; i++) {
    return_p += (double)h_p[i];
  }

  delete [] mat_t;
  delete[] h_p;

  double perman = (4*(nov&1)-2) * return_p;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;


  //return((4*(nov&1)-2) * p);
}

template <class C, class S>
  extern Result gpu_perman64_xshared_coalescing_mshared(DenseMatrix<S>* densemat, flags flags) {
  
  //Pack parameters//
  S* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters//

  //Pack flags//
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;
  int device_id = flags.device_id;
  int grid_dim_multip = flags.grid_multip;
  //Pack flags

  hipSetDevice(device_id);
  hipDeviceSynchronize();

  double starttime = omp_get_wtime();
  
  C x[nov]; 
  C rs; //row sum
  C p = 1; //product of the elements in vector 'x'

  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
      //printf("j: %d -- k: %d \n", j, k);
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //For variable smem
  glob_nov = nov;
  glob_sizeof_c = sizeof(C);
  glob_sizeof_s = sizeof(S);
  //For variable smem
  
  hipOccupancyMaxPotentialBlockSizeVariableSMem(&grid_dim,
                                                 &block_dim,
                                                 &kernel_xshared_coalescing_mshared<C,S>,
                                                 xshared_coalescing_mshared_sharedmem,
                                                 0);

  size_t size = (nov*block_dim*sizeof(C) + nov*nov*sizeof(S));
  
  printf("==SC== Shared memory per block is set to : %zu \n", size);
  printf("==SC== Grid dim is set to : %d \n", grid_dim);
  printf("==SC== Block dim is set to : %d \n", block_dim);
  
  if(grid_dim_multip != 1){
    grid_dim*=grid_dim_multip;
    printf("==SC== Grid dim is re-set to : %d \n", grid_dim);
  }

  S* mat_t = new S[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      //printf("transpose i: %d -- j: %d \n", i, j);
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  S *d_mat_t;
  C *d_x, *d_p;
  C *h_p = new C[grid_dim * block_dim];

  hipMalloc( &d_x, (nov) * sizeof(C));
  hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(C));
  hipMalloc( &d_mat_t, (nov * nov) * sizeof(S));

  hipMemcpy( d_x, x, (nov) * sizeof(C), hipMemcpyHostToDevice);
  hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(S), hipMemcpyHostToDevice);

  long long start = 1;
  long long end = (1LL << (nov-1));

  //double stt = omp_get_wtime();
  kernel_xshared_coalescing_mshared<C,S><<<grid_dim , block_dim , size>>>(d_mat_t, d_x, d_p, nov, start, end);
  hipDeviceSynchronize();
  //double enn = omp_get_wtime();
  //printf("Kernel in %f \n", enn - stt);
  //cout << "kernel" << " in " << (enn - stt) << endl;
  
  hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(C), hipMemcpyDeviceToHost);

  hipFree(d_mat_t);
  hipFree(d_x);
  hipFree(d_p);
  
  //for(int i = 0; i < grid_dim * block_dim; i++){
  //printf("h_p[%d]: %e \n", i, h_p[i]);
  //}

  double return_p = p;
  
  for (int i = 0; i < grid_dim * block_dim; i++) {
    return_p += (double)h_p[i];
    //printf("i: %d -- p: %e  \n", i, p);
  }

  //delete [] mat_t;
  free(mat_t);
  delete[] h_p;

  double perman = (4*(nov&1)-2) * return_p;
  double duration = omp_get_wtime() - starttime;
  Result result(perman, duration);
  return result;

  //return((4*(nov&1)-2) * p);
} 

template <class T>
extern double gpu_perman64_xshared_coalescing_mshared_multigpu(DenseMatrix<T>* densemat, flags flags) {
  
  int gpu_num = flags.gpu_num;
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;

  //Pack parameters
  T* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters
  
  double x[nov]; 
  double rs; //row sum
  double p = 1; //product of the elements in vector 'x'
  double p_partial[gpu_num];
  for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
    p_partial[gpu_id] = 0;
  }
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //create the transpose of the matrix
  T* mat_t = new T[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  long long start = 1;
  long long end = (1LL << (nov-1));
  long long offset = (end - start) / gpu_num;

  #pragma omp parallel for num_threads(gpu_num)
    for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
      hipSetDevice(gpu_id);
      T *d_mat_t;
      double *d_x, *d_p;
      double *h_p = new double[grid_dim * block_dim];

      hipMalloc( &d_x, (nov) * sizeof(double));
      hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(double));
      hipMalloc( &d_mat_t, (nov * nov) * sizeof(T));

      hipMemcpy( d_x, x, (nov) * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(T), hipMemcpyHostToDevice);

      
      int x;
      double stt = omp_get_wtime();
      if (gpu_id == gpu_num-1) {
        //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, (start + gpu_id*offset), end);
	x = 1;
      } else {
        //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, (start + gpu_id*offset), (start + (gpu_id+1)*offset));
	x = 2;
      }
      hipDeviceSynchronize();
      double enn = omp_get_wtime();
      printf("Kernel in %f \n", enn - stt);
      //cout << "kernel" << gpu_id << " in " << (enn - stt) << endl;
        
      hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(double), hipMemcpyDeviceToHost);

      hipFree(d_mat_t);
      hipFree(d_x);
      hipFree(d_p);
      for (int i = 0; i < grid_dim * block_dim; i++) {
        p_partial[gpu_id] += h_p[i];
      }
      delete[] h_p;
    }

  delete [] mat_t;
  for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
    p += p_partial[gpu_id];
  }

  return((4*(nov&1)-2) * p);
}

template <class T>
extern double gpu_perman64_xshared_coalescing_mshared_multigpucpu_chunks(DenseMatrix<T>* densemat, flags flags) {

  
  int gpu_num = flags.gpu_num;
  bool cpu = flags.cpu;
  int threads = flags.threads;
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;

  //Pack parameters
  T* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters
  
  double x[nov]; 
  double rs; //row sum
  double p = 1; //product of the elements in vector 'x'
  double p_partial[gpu_num+1];
  for (int id = 0; id < gpu_num+1; id++) {
    p_partial[id] = 0;
  }

  int number_of_chunks = 1;
  int init = 29;
  if (cpu) {
    init = 28;
  }
  for (int i = init; i < nov; i++) {
    number_of_chunks *= 2;
  }
  int chunk_id = 0;
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //create the transpose of the matrix
  T* mat_t = new T[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  long long start = 1;
  long long end = (1LL << (nov-1));
  long long offset = (end - start) / number_of_chunks;

  omp_set_nested(1);
  omp_set_dynamic(0);
  #pragma omp parallel for num_threads(gpu_num+1)
    for (int id = 0; id < gpu_num+1; id++) {
      if (id == gpu_num) {
        if (cpu) {
          int curr_chunk_id;
          #pragma omp critical 
          {
            curr_chunk_id = chunk_id;
            chunk_id++;
          }
          while (curr_chunk_id < number_of_chunks) {
            double stt = omp_get_wtime();
            if (curr_chunk_id == number_of_chunks - 1) {
              p_partial[id] += cpu_perman64(mat_t, x, nov, (start + curr_chunk_id*offset), end, threads);
            } else {
              p_partial[id] += cpu_perman64(mat_t, x, nov, (start + curr_chunk_id*offset), (start + (curr_chunk_id+1)*offset), threads);
            }
            double enn = omp_get_wtime();
	    printf("ChunkID %d is DONE by CPU in %f \n", curr_chunk_id, enn - stt);
            //cout << "ChunkID " << curr_chunk_id << "is DONE by CPU" << " in " << (enn - stt) << endl;
            #pragma omp critical 
            {
              curr_chunk_id = chunk_id;
              chunk_id++;
            }
          }
        }
      } else {
        hipSetDevice(id);
        
        T *d_mat_t;
        double *d_x, *d_p;
        double *h_p = new double[grid_dim * block_dim];

        hipMalloc( &d_x, (nov) * sizeof(double));
        hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(double));
        hipMalloc( &d_mat_t, (nov * nov) * sizeof(T));

        hipMemcpy( d_x, x, (nov) * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(T), hipMemcpyHostToDevice);

        int curr_chunk_id;
            
        #pragma omp critical 
        {
          curr_chunk_id = chunk_id;
          chunk_id++;
        }
	int x;
        while (curr_chunk_id < number_of_chunks) {
          double stt = omp_get_wtime();
          if (curr_chunk_id == number_of_chunks - 1) {
            //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, (start + curr_chunk_id*offset), end);
	    x = 1;
          } else {
            //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, (start + curr_chunk_id*offset), (start + (curr_chunk_id+1)*offset));
	    x = 2;
          }
          hipDeviceSynchronize();
          double enn = omp_get_wtime();
	  printf("ChunkID %d is DONE by kernel %d in %f \n", curr_chunk_id, id, enn - stt);
          //cout << "ChunkID " << curr_chunk_id << "is DONE by kernel" << id << " in " << (enn - stt) << endl;
	  
          hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(double), hipMemcpyDeviceToHost);
          
          for (int i = 0; i < grid_dim * block_dim; i++) {
            p_partial[id] += h_p[i];
          }
              
          #pragma omp critical 
          {
            curr_chunk_id = chunk_id;
            chunk_id++;
          }
        }

        hipFree(d_mat_t);
        hipFree(d_x);
        hipFree(d_p);
        delete[] h_p;
      }
    }
    
    delete [] mat_t;
    for (int id = 0; id < gpu_num+1; id++) {
      p += p_partial[id];
    }
    
    return((4*(nov&1)-2) * p);
}


template <class T>
extern double gpu_perman64_xshared_coalescing_mshared_multigpu_manual_distribution(DenseMatrix<T>* densemat, flags flags) {

  int gpu_num = flags.gpu_num;
  int grid_dim = flags.grid_dim;
  int block_dim = flags.block_dim;

  //Pack parameters
  T* mat = densemat->mat;
  int nov = densemat->nov;
  //Pack parameters
  
  double x[nov]; 
  double rs; //row sum
  double p = 1; //product of the elements in vector 'x'
  double p_partial[gpu_num];
  for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
    p_partial[gpu_id] = 0;
  }
  
  //create the x vector and initiate the permanent
  for (int j = 0; j < nov; j++) {
    rs = .0f;
    for (int k = 0; k < nov; k++) {
      rs += mat[(j * nov) + k];  // sum of row j
    }
    x[j] = mat[(j * nov) + (nov-1)] - rs/2;  // see Nijenhuis and Wilf - x vector entry
    p *= x[j];   // product of the elements in vector 'x'
  }

  //create the transpose of the matrix
  T* mat_t = new T[nov * nov];
  for (int i = 0; i < nov; i++) {
    for (int j = 0; j < nov; j++) {
      mat_t[(i * nov) + j] = mat[(j * nov) + i];
    }
  }

  long long start = 1;
  long long end = (1LL << (nov-1));
  long long offset = (end - start) / 8;

  #pragma omp parallel for num_threads(gpu_num)
    for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
      hipSetDevice(gpu_id);
      T *d_mat_t;
      double *d_x, *d_p;
      double *h_p = new double[grid_dim * block_dim];

      hipMalloc( &d_x, (nov) * sizeof(double));
      hipMalloc( &d_p, (grid_dim * block_dim) * sizeof(double));
      hipMalloc( &d_mat_t, (nov * nov) * sizeof(T));

      hipMemcpy( d_x, x, (nov) * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy( d_mat_t, mat_t, (nov * nov) * sizeof(T), hipMemcpyHostToDevice);

      int x;
      
      double stt = omp_get_wtime();
      if (gpu_id == 0) {
	//kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, start, start + 3*offset);
	x = 1;
      } else if (gpu_id == 1) {
        //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, start + 3*offset, start + 6*offset);
	x = 2;
      } else if (gpu_id == 2) {
        //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, start + 6*offset, start + 7*offset);
      } else if (gpu_id == 3) {
        //kernel_xshared_coalescing_mshared<<< grid_dim , block_dim , (nov*block_dim*sizeof(float) + nov*nov*sizeof(T)) >>> (d_mat_t, d_x, d_p, nov, start + 7*offset, end);
	x = 3;
      }
      hipDeviceSynchronize();
      double enn = omp_get_wtime();
      printf("Kernel in %f \n", enn - stt);
      //cout << "kernel" << gpu_id << " in " << (enn - stt) << endl;
        
      hipMemcpy( h_p, d_p, grid_dim * block_dim * sizeof(double), hipMemcpyDeviceToHost);

      hipFree(d_mat_t);
      hipFree(d_x);
      hipFree(d_p);
      for (int i = 0; i < grid_dim * block_dim; i++) {
        p_partial[gpu_id] += h_p[i];
      }
      delete[] h_p;
    }

  delete [] mat_t;
  for (int gpu_id = 0; gpu_id < gpu_num; gpu_id++) {
    p += p_partial[gpu_id];
  }

  return((4*(nov&1)-2) * p);
}



//Explicit instantiations required for separate compilation

/////
template extern Result gpu_perman64_xglobal<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xglobal<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xglobal<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xglobal<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xglobal<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_xglobal<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

/////
template extern Result gpu_perman64_xlocal<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xlocal<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xlocal<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xlocal<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xlocal<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_xlocal<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

/////
template extern Result gpu_perman64_xshared<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_xshared<double, double>(DenseMatrix<double>* densemat, flags flags);
/////


/////
template extern Result gpu_perman64_xshared_coalescing<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing<float, double>(DenseMatrix<double>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing<double, double>(DenseMatrix<double>* densemat, flags flags);
/////

/////
template extern Result gpu_perman64_xshared_coalescing_mshared<float, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing_mshared<double, int>(DenseMatrix<int>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing_mshared<float, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing_mshared<double, float>(DenseMatrix<float>* densemat, flags flags);
template extern Result gpu_perman64_xshared_coalescing_mshared<float, double>(DenseMatrix<double>* densemat,flags flags);
template extern Result gpu_perman64_xshared_coalescing_mshared<double, double>(DenseMatrix<double>* densemat,flags flags);
/////


template extern double gpu_perman64_xshared_coalescing_mshared_multigpu<int>(DenseMatrix<int>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpu<float>(DenseMatrix<float>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpu<double>(DenseMatrix<double>* densemat, flags flags);


template extern double gpu_perman64_xshared_coalescing_mshared_multigpucpu_chunks<int>(DenseMatrix<int>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpucpu_chunks<float>(DenseMatrix<float>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpucpu_chunks<double>(DenseMatrix<double>* densemat, flags flags);


template extern double gpu_perman64_xshared_coalescing_mshared_multigpu_manual_distribution<int>(DenseMatrix<int>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpu_manual_distribution<float>(DenseMatrix<float>* densemat, flags flags);
template extern double gpu_perman64_xshared_coalescing_mshared_multigpu_manual_distribution<double>(DenseMatrix<double>* densemat, flags flags);
//Explicit instantiations required for separated compilation
